#include "hip/hip_runtime.h"
#ifndef __CUDALOCKSMUTEXFA_CU__
#define __CUDALOCKSMUTEXFA_CU__

#include "cudaLocks.h"

inline __host__ hipError_t cudaMutexCreateFA(cudaMutex_t * const handle,
                                              const int mutexNumber)
{
  *handle = mutexNumber;
  return hipSuccess;
}

inline __device__ void cudaMutexFALock(const cudaMutex_t mutex,
                                       unsigned int * mutexBufferHeads,
                                       unsigned int * mutexBufferTails,
                                       const int NUM_SM)
{
  const bool isMasterThread = (threadIdx.x == 0 && threadIdx.y == 0 &&
                               threadIdx.z == 0);
  __shared__ unsigned int myTicketNum;
  __shared__ bool haveLock;
  const unsigned int maxTurnNum = 1000000000;

  unsigned int * const ticketNumber = mutexBufferHeads + (mutex * NUM_SM);
  volatile unsigned int * const turnNumber =
      (volatile unsigned int * )mutexBufferTails + (mutex * NUM_SM);

  __syncthreads();
  if (isMasterThread)
  {
    /*
      Use a reprogrammed atomicAnd to get the same functionality as atomicInc
      but without the store release semantics -- the atomicExch determines the
      happens-before ordering here.
    */
    myTicketNum = atomicAnd(ticketNumber, maxTurnNum);
    haveLock = false;
  }
  __syncthreads();
  while (!haveLock)
  {
    if (isMasterThread)
    {
      /*
        Look up the value of turnNumber with an atomic -- use a reprogrammed
        atomicXor (reads value + acquire semantics to invalidate appropriate
        data).
      */
      unsigned int currTicketNum = atomicXor((unsigned int *)turnNumber, 0);

      // it's my turn, I get the lock now
      if (currTicketNum == myTicketNum) { haveLock = true; }
    }
    __syncthreads();
  }
}

inline __device__ void cudaMutexFAUnlock(const cudaMutex_t mutex,
                                         unsigned int * mutexBufferTails,
                                         const int NUM_SM)
{
  const bool isMasterThread = (threadIdx.x == 0 && threadIdx.y == 0 &&
                               threadIdx.z == 0);
  const unsigned int maxTurnNum = 1000000000;
  unsigned int * const turnNumber = mutexBufferTails + (mutex * NUM_SM);

  __syncthreads();
  if (isMasterThread) { atomicInc(turnNumber, maxTurnNum); }
  __syncthreads();
}

// same algorithm but uses per-SM lock
inline __device__ void cudaMutexFALockLocal(const cudaMutex_t mutex,
                                            const unsigned int smID,
                                            unsigned int * mutexBufferHeads,
                                            unsigned int * mutexBufferTails,
                                            const int NUM_SM)
{
  // local variables
  const bool isMasterThread = (threadIdx.x == 0 && threadIdx.y == 0 &&
                               threadIdx.z == 0);
  __shared__ unsigned int myTicketNum;
  __shared__ bool haveLock;
  const unsigned int maxTurnNum = 100000000;

  unsigned int * const ticketNumber = mutexBufferHeads + ((mutex * NUM_SM) +
                                                          smID);
  volatile unsigned int * const turnNumber =
      (volatile unsigned int *)mutexBufferTails + ((mutex * NUM_SM) + smID);

  __syncthreads();
  if (isMasterThread)
  {
    /*
      Use a reprogrammed atomicAnd to get the same functionality as atomicInc
      but without the store release semantics -- the atomicExch determines the
      happens-before ordering here.
    */
    myTicketNum = atomicAnd(ticketNumber, maxTurnNum);
    haveLock = false;
  }
  __syncthreads();
  while (!haveLock)
  {
    if (isMasterThread)
    {
      /*
        Look up the value of turnNumber with an atomic.  Use a reprogrammed
        atomicXor (reads value + acquire semantics to invalidate the appropriate
        data).
      */
      unsigned int currTicketNum = atomicXor((unsigned int *)turnNumber, 0);

      // it's my turn, I get the lock now
      if (currTicketNum == myTicketNum) { haveLock = true; }
    }
    __syncthreads();
  }
}

// same algorithm but uses per-SM lock
inline __device__ void cudaMutexFAUnlockLocal(const cudaMutex_t mutex,
                                              const unsigned int smID,
                                              unsigned int * mutexBufferTails,
                                              const int NUM_SM)
{
  const bool isMasterThread = (threadIdx.x == 0 && threadIdx.y == 0 &&
                               threadIdx.z == 0);
  const unsigned int maxTurnNum = 100000000;

  unsigned int * const turnNumber = mutexBufferTails + ((mutex * NUM_SM) + smID);

  __syncthreads();
  if (isMasterThread) { atomicInc(turnNumber, maxTurnNum); }
  __syncthreads();
}

#endif
